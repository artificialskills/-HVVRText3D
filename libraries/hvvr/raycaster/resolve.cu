#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree. An additional grant
 * of patent rights can be found in the PATENTS file in the same directory.
 */

 //This source code has been partially 
 //modified by Apollo Ellis Jan 2018 and June 2019.

#include "cuda_decl.h"
#include "gbuffer.h"
#include "gpu_camera.h"
#include "gpu_context.h"
#include "kernel_constants.h"
#include "prim_tests.h"
#include "shading.h"
#include "tile_data.h"
#include "warp_ops.h"
#include "grid_buffers.h"

#define PROFILE_RESOLVE 0
#define ENABLE_RESOLVE_STATS 0

namespace hvvr {

extern hipTextureObject_t* gDeviceTextureArray;
extern TextGridGPU gDeviceTextGrid;

struct ResolveStats {
    uint32_t invocations;  // number of pixels shaded, including helpers
    uint32_t shadeSamples; // number samples shaded (invocations * shade count per pixel)
};

struct ResolveSMem {
    ResolveSMem() {}

    union {
        TileData tile;
        TileDataDoF tileDoF;
    };
};

template <uint32_t AARate, uint32_t BlockSize, bool EnableDoF>
CUDA_DEVICE vector4 ShadeSSAA(ResolveSMem& sMem,
                              const RaycasterGBufferSubsample* CUDA_RESTRICT gBufferWarp,
                              int laneIndex,
                              UnpackedDirectionalSample sample3D,
                              vector3 lensCenterToFocalCenter,
                              vector2 frameJitter,
                              const vector2* CUDA_RESTRICT tileSubsampleLensPos,
                              vector3 cameraPos,
                              vector3 cameraLookVector,
                              const PrecomputedTriangleIntersect* CUDA_RESTRICT trianglesIntersect,
                              const PrecomputedTriangleShade* CUDA_RESTRICT trianglesShade,
                              const ShadingVertex* CUDA_RESTRICT verts,
                              const SimpleMaterial* CUDA_RESTRICT materials,
                              hipTextureObject_t* textures,
							  TextGridGPU textGrid,
                              const LightingEnvironment& env,
                              uint32_t sampleOffset,
                              const SampleInfo& sampleInfo,
                              ResolveStats* resolveStats) {
    enum : uint32_t { badTriIndex = ~uint32_t(0) };
    float derivativeMultiplier = rsqrtf(float(AARate));

    vector4 result = vector4(0.0f, 0.0f, 0.0f, 0.0f);
    uint32_t combinedSampleMask = 0;
    for (int compGbufferSlot = 0; compGbufferSlot < AARate; compGbufferSlot++) {
        if (combinedSampleMask == RaycasterGBufferSubsample::getSampleMaskAll<AARate>()) {
            // all samples accounted for, nothing left to shade
            break;
        }

        // GBuffer texels are organized so that each subsample is a warp stride away from
        // the previous subsample for a single sample, so that warps can coalesce memory reads
        uint32_t triIndex = gBufferWarp[compGbufferSlot * WARP_SIZE + laneIndex].triIndex;
        if (triIndex == badTriIndex)
            break; // if we have samples which don't hit a triangle, they will be grouped together as the last entry

        uint32_t sampleMask = gBufferWarp[compGbufferSlot * WARP_SIZE + laneIndex].sampleMask;
        combinedSampleMask |= sampleMask;

        const PrecomputedTriangleIntersect& triIntersect = trianglesIntersect[triIndex];
        const PrecomputedTriangleShade& triShade = trianglesShade[triIndex];

        IntersectTriangleTileDoF triTileDoF;
        triTileDoF.setup(triIntersect, sMem.tileDoF.lensCenter, sMem.tileDoF.lensU, sMem.tileDoF.lensV);
        IntersectTriangleThreadDoF triThreadDoF(triTileDoF, lensCenterToFocalCenter);

        IntersectTriangleTile triTile;
        triTile.setup(triIntersect, sMem.tile.rayOrigin, sMem.tile.majorDirDiff, sMem.tile.minorDirDiff);
        IntersectTriangleThread triThread(triTile, sample3D.centerDir);

        while (sampleMask) {
            int subsampleIndex = __ffs(sampleMask) - 1;
            sampleMask &= ~(1 << subsampleIndex);

#if ENABLE_RESOLVE_STATS
            atomicAdd(&resolveStats->shadeSamples, 1);
#endif

            vector3 b;
            vector3 bOffX;
            vector3 bOffY;
            if (EnableDoF) {
                vector2 lensUV;
                vector2 dirUV;
                GetSampleUVsDoF<AARate, BlockSize>(tileSubsampleLensPos, frameJitter, sMem.tileDoF.focalToLensScale,
                                                   subsampleIndex, lensUV, dirUV);

                triThreadDoF.calcUVW(triTileDoF, lensCenterToFocalCenter, sMem.tileDoF.lensU, sMem.tileDoF.lensV,
                                     lensUV, dirUV, b);

                // we don't have proper derivatives for the DoF path, yet
                vector2 dirUV_dX = dirUV + vector2(sMem.tileDoF.focalToLensScale.x, 0.0f) * derivativeMultiplier;
                triThreadDoF.calcUVW(triTileDoF, lensCenterToFocalCenter, sMem.tileDoF.lensU, sMem.tileDoF.lensV,
                                     lensUV, dirUV_dX, bOffX);

                vector2 dirUV_dY = dirUV + vector2(0.0f, sMem.tileDoF.focalToLensScale.y) * derivativeMultiplier;
                triThreadDoF.calcUVW(triTileDoF, lensCenterToFocalCenter, sMem.tileDoF.lensU, sMem.tileDoF.lensV,
                                     lensUV, dirUV_dY, bOffY);
            } else {
                vector2 alpha = getSubsampleUnitOffset<AARate>(frameJitter, subsampleIndex);

                triThread.calcUVW(triTile, alpha, b);

                vector2 alpha_dX = alpha + vector2(1.0f, 0.0f) * derivativeMultiplier;
                triThread.calcUVW(triTile, alpha_dX, bOffX);

                vector2 alpha_dY = alpha + vector2(0.0f, 1.0f) * derivativeMultiplier;
                triThread.calcUVW(triTile, alpha_dY, bOffY);
            }

            InterpolatedVertex vInterp = interpolate(verts, triShade, b);
            InterpolatedVertex vInterpX = interpolate(verts, triShade, bOffX);
            InterpolatedVertex vInterpY = interpolate(verts, triShade, bOffY);
            vector2 dUVdX = vInterpX.uv - vInterp.uv;
            vector2 dUVdY = vInterpY.uv - vInterp.uv;

#if COLOR_SHADING_MODE == SM_BARYCENTRIC
            result += BarycentricShade(b);
#elif COLOR_SHADING_MODE == SM_TRI_ID
            result += hashedColor(triIndex);
#elif COLOR_SHADING_MODE == SM_UV
            result += UVShade(vInterp);
#elif COLOR_SHADING_MODE == SM_WS_NORMAL
            result += WSNormalShade(vInterp);
#elif COLOR_SHADING_MODE == SM_NO_MATERIAL_BRDF
            result += NoMaterialBRDFShade(vInterp, cameraPos);
#elif COLOR_SHADING_MODE == SM_LAMBERTIAN_TEXTURE
            result += LambertianTextureShade(triShade.material, vInterp, materials, textures);
#elif COLOR_SHADING_MODE == SM_FULL_BRDF
            result += GGXShade(triShade.material, vInterp, dUVdX, dUVdY, cameraPos, materials, textures, textGrid, env);
#elif COLOR_SHADING_MODE == SM_MATERIAL_ID
            result += hashedColor(triShade.material);
#else
#error "NYI"
#endif
        }
    }

    result *= (1.0f / AARate);
    result.w = 1.0f;
    return result;
}

template <uint32_t AARate, uint32_t BlockSize, bool EnableDoF>
CUDA_DEVICE vector4 ShadeMSAA(ResolveSMem& sMem,
                              const RaycasterGBufferSubsample* CUDA_RESTRICT gBufferWarp,
                              int laneIndex,
                              UnpackedDirectionalSample sample3D,
                              vector3 lensCenterToFocalCenter,
                              vector2 frameJitter,
                              const vector2* CUDA_RESTRICT tileSubsampleLensPos,
                              vector3 cameraPos,
                              vector3 cameraLookVector,
                              const PrecomputedTriangleIntersect* CUDA_RESTRICT trianglesIntersect,
                              const PrecomputedTriangleShade* CUDA_RESTRICT trianglesShade,
                              const ShadingVertex* CUDA_RESTRICT verts,
                              const SimpleMaterial* CUDA_RESTRICT materials,
                              hipTextureObject_t* textures,
							  TextGridGPU textGrid,
                              const LightingEnvironment& env,
                              uint32_t sampleOffset,
                              const SampleInfo& sampleInfo,
                              ResolveStats* resolveStats) {
    enum : uint32_t { badTriIndex = ~uint32_t(0) };

    vector4 result = vector4(0.0f, 0.0f, 0.0f, 0.0f);
    uint32_t combinedSampleMask = 0;
    for (int compGbufferSlot = 0; compGbufferSlot < AARate; compGbufferSlot++) {
        if (combinedSampleMask == RaycasterGBufferSubsample::getSampleMaskAll<AARate>()) {
            // all samples accounted for, nothing left to shade
            break;
        }

        // GBuffer texels are organized so that each subsample is a warp stride away from
        // the previous subsample for a single sample, so that warps can coalesce memory reads
        uint32_t triIndex = gBufferWarp[compGbufferSlot * WARP_SIZE + laneIndex].triIndex;
        if (triIndex == badTriIndex)
            break; // if we have samples which don't hit a triangle, they will be grouped together as the last entry

        uint32_t sampleMask = gBufferWarp[compGbufferSlot * WARP_SIZE + laneIndex].sampleMask;
        combinedSampleMask |= sampleMask;

#if ENABLE_RESOLVE_STATS
        atomicAdd(&resolveStats->shadeSamples, 1);
#endif

        float sampleCountInv = 1.0f / __popc(sampleMask);
        // TODO(anankervis): should we always shade from the lens center and center dir (UVs = 0)?
        vector2 centroidLensUV = vector2(0.0f, 0.0f);
        vector2 centroidDirUV = vector2(0.0f, 0.0f);
        // Is it correct that we're generating the centroid by averaging all sample positions?
        // Typically, you'd want the centroid to be the sample (not subsample) center, and only fall back to
        // something else (clamp to tri edge?) if the sample center falls outside the triangle.
        // Now that we've got DoF samples which vary their distribution from pixel to pixel, it's even
        // more important that we pick a consistent sampling location, to avoid wavy artifacts.
        vector2 centroidAlpha = vector2(0.0f, 0.0f);
        for (uint32_t centroidMask = sampleMask; centroidMask != 0;) {
            int subsampleIndex = __ffs(centroidMask) - 1;
            centroidMask &= ~(1 << subsampleIndex);
            centroidAlpha += getSubsampleUnitOffset<AARate>(frameJitter, subsampleIndex) * sampleCountInv;

            vector2 lensUV;
            vector2 dirUV;
            GetSampleUVsDoF<AARate, BlockSize>(tileSubsampleLensPos, frameJitter, sMem.tileDoF.focalToLensScale,
                                               subsampleIndex, lensUV, dirUV);
            centroidLensUV += lensUV * sampleCountInv;
            centroidDirUV += dirUV * sampleCountInv;
        }

        const PrecomputedTriangleIntersect& triIntersect = trianglesIntersect[triIndex];
        const PrecomputedTriangleShade& triShade = trianglesShade[triIndex];

        IntersectTriangleTileDoF triTileDoF;
        triTileDoF.setup(triIntersect, sMem.tileDoF.lensCenter, sMem.tileDoF.lensU, sMem.tileDoF.lensV);
        IntersectTriangleThreadDoF triThreadDoF(triTileDoF, lensCenterToFocalCenter);

        IntersectTriangleTile triTile;
        triTile.setup(triIntersect, sMem.tile.rayOrigin, sMem.tile.majorDirDiff, sMem.tile.minorDirDiff);
        IntersectTriangleThread triThread(triTile, sample3D.centerDir);

        vector3 b;
        vector3 bOffX;
        vector3 bOffY;
        if (EnableDoF) {
            triThreadDoF.calcUVW(triTileDoF, lensCenterToFocalCenter, sMem.tileDoF.lensU, sMem.tileDoF.lensV,
                                 centroidLensUV, centroidDirUV, b);

            // TODO(anankervis): we don't have proper derivatives for the DoF path, yet
            // but this seems to work well when the lens radius is relatively small and the focal plane is close
            // if the focal plane is too far, the derivatives become too small and you get aliasing
            // if the lens is too big, the derivatives are too large and things are always blurry
            // There's implicit scaling due to lensCenterToFocalCenter (focalDist) and lensU/lensV (lensRadius) -
            // see calcUVW and TileDataDoF::load.
            // Biasing mipmap selection will never be as good of quality as SSAA (due to bilinear square-shaped filter),
            // but it just needs to be reasonably close because we're not trying to simulate very wide filters
            // in the general case. It only needs to be convincing for relatively small filters.
            vector2 derivScale = vector2(1.0f, 1.0f);

            vector2 dirUV_dX = centroidDirUV + vector2(derivScale.x, 0.0f);
            triThreadDoF.calcUVW(triTileDoF, lensCenterToFocalCenter, sMem.tileDoF.lensU, sMem.tileDoF.lensV,
                                 centroidLensUV, dirUV_dX, bOffX);

            vector2 dirUV_dY = centroidDirUV + vector2(0.0f, derivScale.y);
            triThreadDoF.calcUVW(triTileDoF, lensCenterToFocalCenter, sMem.tileDoF.lensU, sMem.tileDoF.lensV,
                                 centroidLensUV, dirUV_dY, bOffY);
        } else {
            triThread.calcUVW(triTile, centroidAlpha, b);

            vector2 alpha_dX = centroidAlpha + vector2(1.0f, 0.0f);
            triThread.calcUVW(triTile, alpha_dX, bOffX);

            vector2 alpha_dY = centroidAlpha + vector2(0.0f, 1.0f);
            triThread.calcUVW(triTile, alpha_dY, bOffY);
        }

        InterpolatedVertex vInterp = interpolate(verts, triShade, b);
        InterpolatedVertex vInterpX = interpolate(verts, triShade, bOffX);
        InterpolatedVertex vInterpY = interpolate(verts, triShade, bOffY);
        vector2 dUVdX = vInterpX.uv - vInterp.uv;
        vector2 dUVdY = vInterpY.uv - vInterp.uv;

        vector4 shadedColor = GGXShade(triShade.material, vInterp, dUVdX, dUVdY, cameraPos, materials, textures, textGrid, env);
        result += shadedColor * __popc(sampleMask);
    }
    result *= 1.0f / AARate;
    result.w = 1.0f;
    return result;
}

template <uint32_t AARate, uint32_t BlockSize, bool EnableDoF>
CUDA_DEVICE vector4 ShadeAndResolve(ResolveSMem& sMem,
                                    const RaycasterGBufferSubsample* CUDA_RESTRICT gBufferBlock,
                                    int laneIndex,
                                    uint32_t sampleOffset,
                                    SampleInfo sampleInfo,
                                    UnpackedDirectionalSample sample3D,
                                    vector3 lensCenterToFocalCenter,
                                    const vector2* CUDA_RESTRICT tileSubsampleLensPos,
                                    vector3 cameraPos,
                                    vector3 cameraLookVector,
                                    const PrecomputedTriangleIntersect* CUDA_RESTRICT trianglesIntersect,
                                    const PrecomputedTriangleShade* CUDA_RESTRICT trianglesShade,
                                    const ShadingVertex* CUDA_RESTRICT verts,
                                    const SimpleMaterial* CUDA_RESTRICT materials,
                                    hipTextureObject_t* textures,
									TextGridGPU textGrid,
                                    LightingEnvironment env,
                                    ResolveStats* resolveStats) {
#if ENABLE_RESOLVE_STATS
    atomicAdd(&resolveStats->invocations, 1);
#endif

    vector4 result =
#if SUPERSHADING_MODE == SSAA_SHADE
        ShadeSSAA<AARate, BlockSize, EnableDoF>(sMem, gBufferBlock, laneIndex, sample3D, lensCenterToFocalCenter,
                                                sampleInfo.frameJitter, tileSubsampleLensPos, cameraPos,
                                                cameraLookVector, trianglesIntersect, trianglesShade, verts, materials,
                                                textures, env, sampleOffset, sampleInfo, resolveStats);
#else
        ShadeMSAA<AARate, BlockSize, EnableDoF>(sMem, gBufferBlock, laneIndex, sample3D, lensCenterToFocalCenter,
                                                sampleInfo.frameJitter, tileSubsampleLensPos, cameraPos,
                                                cameraLookVector, trianglesIntersect, trianglesShade, verts, materials,
                                                textures, textGrid, env, sampleOffset, sampleInfo, resolveStats);
#endif

    return result;
}

template <uint32_t AARate, uint32_t BlockSize, bool TMaxBuffer, bool EnableDoF>
CUDA_KERNEL void ResolveKernel(uint32_t* sampleResults,
                               float* tMaxBuffer,
                               const RaycasterGBufferSubsample* CUDA_RESTRICT gBuffer,
                               SampleInfo sampleInfo,
                               matrix4x4 sampleToWorld,
                               matrix3x3 sampleToCamera,
                               matrix4x4 cameraToWorld,
                               const vector2* CUDA_RESTRICT tileSubsampleLensPos,
                               const unsigned* CUDA_RESTRICT tileIndexRemapOccupied,
                               vector3 cameraPos,
                               vector3 cameraLookVector,
                               const PrecomputedTriangleIntersect* CUDA_RESTRICT trianglesIntersect,
                               const PrecomputedTriangleShade* CUDA_RESTRICT trianglesShade,
                               const ShadingVertex* CUDA_RESTRICT verts,
                               const SimpleMaterial* CUDA_RESTRICT materials,
                               hipTextureObject_t* textures,
							   TextGridGPU textGrid,
                               LightingEnvironment env,
                               ResolveStats* resolveStats) {
    static_assert(TILE_SIZE == BlockSize, "ResolveKernel assumes TILE_SIZE == BlockSize");

    int laneIndex = laneGetIndex();

    uint32_t rayInTileIndex = threadIdx.x;
    uint32_t compactedTileIndex = blockIdx.x;
    uint32_t tileIndex = tileIndexRemapOccupied[compactedTileIndex];
    uint32_t sampleOffset = tileIndex * TILE_SIZE + rayInTileIndex;

    // GBuffer texels are organized so that each subsample is a warp stride away from
    // the previous subsample for a single sample, so that warps can coalesce memory reads
    uint32_t warpIndex = sampleOffset / WARP_SIZE;
    uint32_t warpOffset = warpIndex * WARP_SIZE * AARate;

    UnpackedDirectionalSample sample3D =
        GetDirectionalSample3D(sampleOffset, sampleInfo, sampleToWorld, sampleToCamera, cameraToWorld);

    UnpackedSample sample2D = GetFullSample(sampleOffset, sampleInfo);
    matrix3x3 sampleToWorldRotation = matrix3x3(sampleToWorld);
    vector3 lensCenterToFocalCenter =
        sampleInfo.lens.focalDistance * (sampleToWorldRotation * vector3(sample2D.center.x, sample2D.center.y, 1.0f));

    // TODO(anankervis): precompute this with more accurate values, and load from a per-tile buffer
    // (but watch out for the foveated path)
    __shared__ ResolveSMem sMem;
    if (threadIdx.x == BlockSize / 2) {
        if (EnableDoF) {
            sMem.tileDoF.load(sampleInfo, sampleToWorld, sampleOffset);
        } else {
            sMem.tile.load(sampleToWorld, sample3D);
        }
    }
    __syncthreads();

    vector4 result = ShadeAndResolve<AARate, BlockSize, EnableDoF>(
        sMem, gBuffer + warpOffset, laneGetIndex(), sampleOffset, sampleInfo, sample3D, lensCenterToFocalCenter,
        tileSubsampleLensPos, cameraPos, cameraLookVector, trianglesIntersect, trianglesShade, verts, materials,
        textures, textGrid, env, resolveStats);

    result = ACESFilm(result);
    sampleResults[sampleOffset] = ToColor4Unorm8SRgb(result);

    if (TMaxBuffer) {
        enum { tMaxSubsampleIndex = 0 };
        vector2 alpha = getSubsampleUnitOffset<AARate>(sampleInfo.frameJitter, tMaxSubsampleIndex);

        // scan through the compressed gbuffer until we find the subsample we care about
        enum : uint32_t { badTriIndex = ~uint32_t(0) };
        float tMaxValue = CUDA_INF;
        uint32_t combinedSampleMask = 0;
        for (int compGbufferSlot = 0; compGbufferSlot < AARate; compGbufferSlot++) {
            if (combinedSampleMask == RaycasterGBufferSubsample::getSampleMaskAll<AARate>()) {
                // all samples accounted for, nothing left to shade
                break;
            }

            // GBuffer texels are organized so that each subsample is a warp stride away from
            // the previous subsample for a single sample, so that warps can coalesce memory reads
            uint32_t triIndex = gBuffer[warpOffset + compGbufferSlot * WARP_SIZE + laneIndex].triIndex;
            if (triIndex == badTriIndex)
                break; // if we have samples which don't hit a triangle, they will be grouped together as the last entry

            uint32_t sampleMask = gBuffer[warpOffset + compGbufferSlot * WARP_SIZE + laneIndex].sampleMask;
            combinedSampleMask |= sampleMask;

            if ((sampleMask & (1 << tMaxSubsampleIndex)) != 0) {
                PrecomputedTriangleIntersect triIntersect = trianglesIntersect[triIndex];

                if (EnableDoF) {
                    IntersectTriangleTileDoF triTileDoF;
                    triTileDoF.setup(triIntersect, sMem.tileDoF.lensCenter, sMem.tileDoF.lensU, sMem.tileDoF.lensV);
                    IntersectTriangleThreadDoF triThreadDoF(triTileDoF, lensCenterToFocalCenter);

                    // should lensUV be forced to zero (centered)?
                    vector2 lensUV;
                    vector2 dirUV;
                    GetSampleUVsDoF<AARate, BlockSize>(tileSubsampleLensPos, sampleInfo.frameJitter,
                                                       sMem.tileDoF.focalToLensScale, tMaxSubsampleIndex, lensUV,
                                                       dirUV);

                    vector3 uvw;
                    triThreadDoF.calcUVW(triTileDoF, lensCenterToFocalCenter, sMem.tileDoF.lensU, sMem.tileDoF.lensV,
                                         lensUV, dirUV, uvw);

                    vector3 v0 = triIntersect.v0;
                    vector3 v1 = triIntersect.v0 + triIntersect.edge0;
                    vector3 v2 = triIntersect.v0 + triIntersect.edge1;

                    vector3 pos = uvw.x * v0 + uvw.y * v1 + uvw.z * v2;

                    vector3 posDelta = pos - cameraPos;
                    tMaxValue = dot(posDelta, cameraLookVector);
                } else {
                    IntersectTriangleTile triTile;
                    triTile.setup(triIntersect, sMem.tile.rayOrigin, sMem.tile.majorDirDiff, sMem.tile.minorDirDiff);
                    IntersectTriangleThread triThread(triTile, sample3D.centerDir);

                    vector3 uvw;
                    triThread.calcUVW(triTile, alpha, uvw);

                    vector3 v0 = triIntersect.v0;
                    vector3 v1 = triIntersect.v0 + triIntersect.edge0;
                    vector3 v2 = triIntersect.v0 + triIntersect.edge1;

                    vector3 pos = uvw.x * v0 + uvw.y * v1 + uvw.z * v2;

                    vector3 posDelta = pos - cameraPos;
                    tMaxValue = dot(posDelta, cameraLookVector);
                }
                break;
            }
        }
        tMaxBuffer[sampleOffset] = tMaxValue;
    }
}

void GPUCamera::shadeAndResolve(GPUSceneState& sceneState, const SampleInfo& sampleInfo) {
    Camera_StreamedData& streamedData = streamed[streamedIndexGPU];

    static_assert(TILE_SIZE % WARP_SIZE == 0, "Tile size must be a multiple of warp size in the current architecture. "
                                              "The 'GBuffer' is interleaved in a way that would break otherwise.");

    ResolveStats* resolveStatsPtr = nullptr;
#if ENABLE_RESOLVE_STATS
    static GPUBuffer<ResolveStats> resolveStatsBuffer(1);
    resolveStatsBuffer.memsetAsync(0, stream);
    resolveStatsPtr = resolveStatsBuffer.data();
#endif

#if PROFILE_RESOLVE
    static uint64_t frameIndex = 0;
    enum { profileFrameSkip = 64 };
    static hipEvent_t start = nullptr;
    static hipEvent_t stop = nullptr;
    static float minTimeMs = FLT_MAX;
    if (!start) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }
    if (frameIndex % profileFrameSkip == 0) {
        hipEventRecord(start, stream);
    }
#endif

#define RESOLVE_LAUNCH(AARate, BlockSize, TMaxBuffer, EnableDoF, dim, stream)                                          \
    ResolveKernel<AARate, BlockSize, TMaxBuffer, EnableDoF><<<dim.grid, dim.block, 0, stream>>>(                       \
        d_sampleResults, d_tMaxBuffer, d_gBuffer, sampleInfo, cameraToWorld * matrix4x4(sampleToCamera),               \
        sampleToCamera, cameraToWorld, d_tileSubsampleLensPos, local.tileIndexRemapOccupied.data(), position,          \
        lookVector, sceneState.trianglesIntersect, sceneState.trianglesShade, sceneState.worldSpaceVertices,           \
        sceneState.materials, gDeviceTextureArray, gDeviceTextGrid, sceneState.lightingEnvironment, resolveStatsPtr)

    KernelDim dimResolve(streamedData.tileCountOccupied * TILE_SIZE, TILE_SIZE);
    if (d_tMaxBuffer.size() != 0) {
        // output a tMax depth buffer for reprojection
        if (sampleInfo.lens.radius > 0.0f) {
            // Enable depth of field
            RESOLVE_LAUNCH(COLOR_MODE_MSAA_RATE, TILE_SIZE, true, true, dimResolve, stream);
        } else {
            // No depth of field, assume all rays have the same origin
            RESOLVE_LAUNCH(COLOR_MODE_MSAA_RATE, TILE_SIZE, true, false, dimResolve, stream);
        }
    } else {
        if (sampleInfo.lens.radius > 0.0f) {
            // Enable depth of field
            RESOLVE_LAUNCH(COLOR_MODE_MSAA_RATE, TILE_SIZE, false, true, dimResolve, stream);
        } else {
            // No depth of field, assume all rays have the same origin
            RESOLVE_LAUNCH(COLOR_MODE_MSAA_RATE, TILE_SIZE, false, false, dimResolve, stream);
        }
    }
#undef RESOLVE_LAUNCH

#if PROFILE_RESOLVE
    if (frameIndex % profileFrameSkip == 0) {
        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);
        float timeMs = 0.0f;
        hipEventElapsedTime(&timeMs, start, stop);
        minTimeMs = min(minTimeMs, timeMs);
        printf("resolve min: %.2fms\n", minTimeMs);
    }
    frameIndex++;
#endif

#if ENABLE_RESOLVE_STATS
    ResolveStats resolveStats = {};
    resolveStatsBuffer.readback(&resolveStats);
    printf("resolve stats: %u invocations, %u shade samples\n", resolveStats.invocations, resolveStats.shadeSamples);
#endif
}

template <bool TMaxBuffer>
CUDA_KERNEL void ClearEmptyKernel(uint32_t* sampleResults,
                                  float* tMaxBuffer,
                                  const uint32_t* CUDA_RESTRICT tileIndexRemapEmpty,
                                  uint32_t emptyTileCount) {
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t compactedTileIndex = index / TILE_SIZE;
    uint32_t threadIndex = index - compactedTileIndex * TILE_SIZE;
    if (compactedTileIndex < emptyTileCount) {
        uint32_t tileIndex = tileIndexRemapEmpty[compactedTileIndex];
        uint32_t sampleOffset = tileIndex * TILE_SIZE + threadIndex;
        sampleResults[sampleOffset] = 0xFF000000;
        if (TMaxBuffer) {
            tMaxBuffer[sampleOffset] = CUDA_INF;
        }
    }
}

void GPUCamera::clearEmpty() {
    Camera_StreamedData& streamedData = streamed[streamedIndexGPU];

    uint32_t tileCount = streamedData.tileCountEmpty;
    uint32_t blockCount = (tileCount * TILE_SIZE + CUDA_GROUP_SIZE - 1) / CUDA_GROUP_SIZE;
    uint32_t* d_emptyTileIndexRemap = local.tileIndexRemapEmpty.data();

    dim3 dimGrid(blockCount, 1, 1);
    dim3 dimBlock(CUDA_GROUP_SIZE, 1, 1);

    if (d_tMaxBuffer.size() != 0) {
        ClearEmptyKernel<true>
            <<<dimGrid, dimBlock, 0, stream>>>(d_sampleResults, d_tMaxBuffer, d_emptyTileIndexRemap, tileCount);
    } else {
        ClearEmptyKernel<false>
            <<<dimGrid, dimBlock, 0, stream>>>(d_sampleResults, nullptr, d_emptyTileIndexRemap, tileCount);
    }
}

} // namespace hvvr
