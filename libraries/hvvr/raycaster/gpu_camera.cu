/**
 * Copyright (c) 2017-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree. An additional grant
 * of patent rights can be found in the PATENTS file in the same directory.
 */

#include "constants_math.h"
#include "gpu_camera.h"
#include "gpu_context.h"
#include "gpu_samples.h"
#include "kernel_constants.h"
#include "magic_constants.h"
#include "memory_helpers.h"

#include <functional>
#include <random>

#ifdef DX_SUPPORTED
#include <cuda_d3d11_interop.h>
#else
struct ID3D11Texture2D;
struct ID3D11Buffer;
#endif

#include <cuda_gl_interop.h>


namespace hvvr {

uint32_t pixelFormatSize(PixelFormat pixelFormat) {
    switch (pixelFormat) {
        case PixelFormat::RGBA8_SRGB:
            return 4;
        case PixelFormat::RGBA32F:
            return 16;
        default:
            assert(false);
            return 0;
    }
}

void Camera_StreamedData::reset(uint32_t tileCount) {
    tileCountOccupied = 0;
    tileCountEmpty = 0;

    tileIndexRemapEmpty.resizeDestructive(tileCount);
    tileIndexRemapOccupied.resizeDestructive(tileCount);
    tileTriRanges.resizeDestructive(tileCount);
    triIndices.resizeDestructive(MAX_TRI_INDICES_TO_INTERSECT);
    tileFrusta3D.resizeDestructive(tileCount);
}

GPUCamera::GPUCamera(const Camera* cameraPtr) : streamedIndexCPU(0), streamedIndexGPU(-1), cameraPtr(cameraPtr) {
    cutilSafeCall(hipStreamCreate(&stream));

    for (int n = 0; n < frameBuffering; n++) {
        cutilSafeCall(hipEventCreateWithFlags(&streamed[n].gpuDone, hipEventBlockingSync | hipEventDisableTiming));
    }
}
// TODO: there's no cleanup code for GPUCamera, yet, and it would be a big pain to clean it up to properly support
// the full set of constructors and assignments (especially move variants) given the number of members...

void GPUCamera::initLookupTables(int _MSAARate) {
    // getSubsampleUnitOffset needs a compile-time constant for MSAARate
    enum { MSAARate = COLOR_MODE_MSAA_RATE };
    assert(MSAARate == _MSAARate);

    std::uniform_real_distribution<float> uniformRandomDist(0.0f, 1.0f);
    std::mt19937 generator;
    auto r = std::bind(uniformRandomDist, std::ref(generator));

    // lookup table for random lens position
    enum { TileCount = DOF_LENS_POS_LOOKUP_TABLE_TILES };
    std::vector<vector2> tileSubsampleLensPosData(TILE_SIZE * TileCount * MSAARate);
    for (int tile = 0; tile < TileCount; tile++) {
        for (int sample = 0; sample < int(TILE_SIZE); sample++) {
            float rotation = r() * Tau;

            for (int subsample = 0; subsample < MSAARate; subsample++) {
                vector2 pos =
                    getSubsampleUnitOffset<MSAARate>(vector2(0.0f, 0.0f), (subsample * 7 + 7) % MSAARate, rotation);

                // tileSubsampleLensPosData[tile * TILE_SIZE * MSAARate + subsample * TILE_SIZE + sample].x =
                //    uint32_t(floatToHalf(pos.x)) | (uint32_t(floatToHalf(pos.y)) << 16);
                tileSubsampleLensPosData[tile * TILE_SIZE * MSAARate + subsample * TILE_SIZE + sample] = pos;
            }
         }
    }
    d_tileSubsampleLensPos.resizeDestructive(TILE_SIZE * TileCount * MSAARate);
    d_tileSubsampleLensPos.upload(tileSubsampleLensPosData.data());
}

Camera_StreamedData* GPUCamera::streamedDataLock(uint32_t tileCount) {
    Camera_StreamedData* rval = streamed + streamedIndexCPU;
    cutilSafeCall(hipEventSynchronize(rval->gpuDone));
	
    streamedIndexCPU = (streamedIndexCPU + 1) % frameBuffering;

    rval->reset(tileCount);
    return rval;
}

void GPUCamera::streamedDataUnlock() {
    streamedIndexGPU = (streamedIndexGPU + 1) % frameBuffering;

    Camera_StreamedData* streamSrc = streamed + streamedIndexGPU;

    // some things don't have appropriate access patterns for reasonable PCIe streaming perf, so we copy them
    local.tileIndexRemapEmpty.resizeDestructive(streamSrc->tileIndexRemapEmpty.size());
    local.tileIndexRemapEmpty.uploadAsync(streamSrc->tileIndexRemapEmpty.data(), stream);

    local.tileIndexRemapOccupied.resizeDestructive(streamSrc->tileIndexRemapOccupied.size());
    local.tileIndexRemapOccupied.uploadAsync(streamSrc->tileIndexRemapOccupied.data(), stream);

    cutilFlush(stream);

    local.tileTriRanges.resizeDestructive(streamSrc->tileTriRanges.size());
    local.tileTriRanges.uploadAsync(streamSrc->tileTriRanges.data(), stream);

    local.tileFrusta3D.resizeDestructive(streamSrc->tileFrusta3D.size());
    local.tileFrusta3D.uploadAsync(streamSrc->tileFrusta3D.data(), stream);

    cutilFlush(stream);
}

void GPUCamera::streamedDataGpuDone() {
    cutilSafeCall(hipEventRecord(streamed[streamedIndexGPU].gpuDone, stream));
    cutilFlush(stream);
}

void GPUCamera::setCameraJitter(vector2 jitter) {
    frameJitter.x = jitter.x;
    frameJitter.y = jitter.y;
}

void GPUCamera::updatePerFrame(vector3 cameraPos,
                               vector3 cameraLookVector,
                               const matrix3x3& _sampleToCamera,
                               const matrix4x4& _cameraToWorld) {
    position = vector3(cameraPos.x, cameraPos.y, cameraPos.z);
    lookVector = vector3(cameraLookVector.x, cameraLookVector.y, cameraLookVector.z);
    sampleToCamera = _sampleToCamera;
    cameraToWorld = _cameraToWorld;
}

static int getMSAARate(RaycasterOutputMode outputMode) {
    return (outputMode == RaycasterOutputMode::COLOR_RGBA8) ? COLOR_MODE_MSAA_RATE : 1;
}

static TextureFormat pixelFormatToTextureFormat(PixelFormat format) {
    switch (format) {
        case PixelFormat::RGBA8_SRGB:
            return TextureFormat::r8g8b8a8_unorm_srgb;
        case PixelFormat::RGBA32F:
            return TextureFormat::r32g32b32a32_float;
        default:
            assert(false);
    }
    return TextureFormat::none;
}

// TODO(anankervis): merge the different functions that duplicate camera resource creation
void GPUCamera::updateConfig(RaycasterOutputMode _outputMode,
                             int32_t* sampleRemap,
                             float* sampleLocations,
                             Sample::Extents* sampleExtents,
                             ThinLens _lens,
                             uint32_t _sampleCount,
                             uint32_t imageWidth,
                             uint32_t imageHeight,
                             uint32_t imageStride,
                             uint32_t _splitColorSamples) {
    splitColorSamples = _splitColorSamples;
    // one sample per output pixel, one sample per pentile subpixel, or one sample per R,G,B channel
    assert(splitColorSamples == 1 || splitColorSamples == 2 || splitColorSamples == 3);

    validSampleCount = imageWidth * imageHeight * splitColorSamples;
    d_sampleRemap = GPUBuffer<int32_t>(sampleRemap, sampleRemap + validSampleCount);
    sampleCount = _sampleCount;
    d_sampleLocations = GPUBuffer<vector2>((vector2*)sampleLocations, (vector2*)(sampleLocations) + sampleCount);
    d_sampleExtents =
        GPUBuffer<Sample::Extents>((Sample::Extents*)sampleExtents, (Sample::Extents*)(sampleExtents) + sampleCount);

    outputMode = _outputMode;
    int msaaRate = getMSAARate(outputMode);
    d_gBuffer = GPUBuffer<RaycasterGBufferSubsample>(sampleCount * msaaRate);

    PixelFormat outputFormat = outputModeToPixelFormat(outputMode);
    TextureFormat textureFormat = pixelFormatToTextureFormat(outputFormat);

    previousResultTexture =
        createEmptyTexture(imageWidth, imageHeight, textureFormat, hipAddressModeClamp, hipAddressModeClamp);
    resultTexture =
        createEmptyTexture(imageWidth, imageHeight, textureFormat, hipAddressModeClamp, hipAddressModeClamp);

    contrastEnhancementSettings.enable = true;
    contrastEnhancementSettings.f_e = 0.2f;
    contrastEnhancementBuffers.horizontallyFiltered =
        createEmptyTexture(imageWidth, imageHeight, textureFormat, hipAddressModeClamp, hipAddressModeClamp);
    contrastEnhancementBuffers.fullyFiltered =
        createEmptyTexture(imageWidth, imageHeight, textureFormat, hipAddressModeClamp, hipAddressModeClamp);

    auto pixelFormat = outputModeToPixelFormat(outputMode);
    d_sampleResults =
        GPUBuffer<uint32_t>((sampleCount * pixelFormatSize(pixelFormat) + sizeof(uint32_t) - 1) / sizeof(uint32_t));
    resultImage.update(imageWidth, imageHeight, imageStride, pixelFormat);
    lens = _lens;

    initLookupTables(msaaRate);
}

void GPUCamera::registerPolarFoveatedSamples(const std::vector<vector2ui>& polarRemapToPixel,
                                             float _maxEccentricityRadians,
                                             const std::vector<float>& ringEccentricities,
                                             const std::vector<float>& eccentricityCoordinateMap,
                                             uint32_t samplesPerRing,
                                             uint32_t paddedSampleCount) {
    PixelFormat outputFormat = outputModeToPixelFormat(outputMode);
    sampleCount = paddedSampleCount;
    d_sampleResults = GPUBuffer<uint32_t>((paddedSampleCount * pixelFormatSize(outputFormat) + sizeof(uint32_t) - 1) /
                                          sizeof(uint32_t));
    d_sampleLocations = GPUBuffer<vector2>(paddedSampleCount);
    d_sampleExtents = GPUBuffer<Sample::Extents>(paddedSampleCount);
    d_sampleRemap = GPUBuffer<int32_t>(paddedSampleCount);

    // For temporal filtering
    d_tMaxBuffer = GPUBuffer<float>(paddedSampleCount);

    maxEccentricityRadians = _maxEccentricityRadians;
    d_eccentricityCoordinateMap = makeGPUBuffer(eccentricityCoordinateMap);
    d_ringEccentricities = makeGPUBuffer(ringEccentricities);
	
    int msaaRate = getMSAARate(outputMode);
    size_t totalSubsampleCount = paddedSampleCount * msaaRate;

    // Allow us to launch a complete tile
    d_gBuffer = GPUBuffer<RaycasterGBufferSubsample>(totalSubsampleCount);

    d_polarRemapToPixel = makeGPUBuffer(polarRemapToPixel);

    TextureFormat textureFormat = pixelFormatToTextureFormat(outputFormat);

    polarFoveatedImage = createEmptyTexture(samplesPerRing, uint32_t(polarRemapToPixel.size() / samplesPerRing),
                                            textureFormat, hipAddressModeWrap, hipAddressModeClamp);
    previousPolarFoveatedImage = createEmptyTexture(samplesPerRing, uint32_t(polarRemapToPixel.size() / samplesPerRing),
                                                    textureFormat, hipAddressModeWrap, hipAddressModeClamp);
    rawPolarFoveatedImage = createEmptyTexture(samplesPerRing, uint32_t(polarRemapToPixel.size() / samplesPerRing),
                                               textureFormat, hipAddressModeWrap, hipAddressModeClamp, false);
    polarFoveatedDepthImage =
        createEmptyTexture(samplesPerRing, uint32_t(polarRemapToPixel.size() / samplesPerRing),
                           TextureFormat::r32_float, hipAddressModeWrap, hipAddressModeClamp, false);

    initLookupTables(msaaRate);
}

bool GPUCamera::bindTexture(GPUContext& gpuContext, ImageResourceDescriptor texture) {
    if (resultsResource) {
        gpuContext.interopUnmapResources();
        cutilSafeCall(hipGraphicsUnregisterResource(resultsResource));
        resultsResource = nullptr;
    }
#ifdef DX_SUPPORTED
    if (texture.memoryType == ImageResourceDescriptor::MemoryType::DX_TEXTURE) {
        // hipGraphicsRegisterFlagsNone is only valid flag as of 7/22/2016
        cutilSafeCall(cudaGraphicsD3D11RegisterResource(&resultsResource, (ID3D11Texture2D*)texture.data,
                                                        hipGraphicsRegisterFlagsNone));
    }
#endif
    if (texture.memoryType == ImageResourceDescriptor::MemoryType::OPENGL_TEXTURE) {
        cutilSafeCall(hipGraphicsGLRegisterImage(&resultsResource, (GLuint)(uint64_t)texture.data, GL_TEXTURE_2D,
                                                  cudaGraphicsMapFlagsWriteDiscard));
    }
	
    return true;
}

void GPUCamera::copyImageToBoundTexture() {
    hipArray* cuArray;
    cutilSafeCall(hipGraphicsSubResourceGetMappedArray(&cuArray, resultsResource, 0, 0));
    size_t srcStride = resultImage.width() * resultImage.bytesPerPixel(); // tightly packed
    cutilSafeCall(hipMemcpy2DToArrayAsync(cuArray, 0, 0, resultImage.data(), srcStride, srcStride,
                                           resultImage.height(), hipMemcpyDeviceToDevice, stream));
}

void GPUCamera::copyImageToCPU(uint32_t* imageData, uint32_t imageWidth, uint32_t imageHeight, uint32_t imageStride) {
    auto pixFormat = outputModeToPixelFormat(outputMode);
    resultImage.update(imageWidth, imageHeight, imageStride, pixFormat);

    cutilSafeCall(
        hipMemcpyAsync(imageData, resultImage.data(), resultImage.sizeInMemory(), hipMemcpyDeviceToHost, 0));
}

void GPUCamera::acquireTileCullData(SimpleRayFrustum* tileFrusta, SimpleRayFrustum* blockFrusta) {
    cutilSafeCall(hipEventSynchronize(transferTileToCPUEvent));

    size_t blockCount = d_cullBlockFrusta.size();
    memcpy(blockFrusta, foveatedWorldSpaceBlockFrustaPinned, sizeof(SimpleRayFrustum) * blockCount);
    memcpy(tileFrusta, foveatedWorldSpaceTileFrustaPinned, sizeof(SimpleRayFrustum) * blockCount * TILES_PER_BLOCK);
}

void GPUCamera::intersectShadeResolve(GPUSceneState& sceneState) {
    Camera_StreamedData& streamedData = streamed[streamedIndexGPU];

    // prep the scene
    sceneState.update();
    cutilSafeCall(hipStreamWaitEvent(stream, sceneState.updateEvent, 0));

    // The intersect and resolve kernels assume every thread will map to a valid work item, with valid input and output
    // slots. Sample count should be padded to a minimum of CUDA_GROUP_SIZE. In practice, it is padded to BLOCK_SIZE.
    assert(sampleCount % CUDA_GROUP_SIZE == 0);

    if (streamedData.tileCountEmpty > 0) {
        clearEmpty();
    }

    SampleInfo sampleInfo(*this);
    if (streamedData.tileCountOccupied > 0) {
        intersect(sceneState, sampleInfo);
        shadeAndResolve(sceneState, sampleInfo);
    }

    streamedDataGpuDone();
}

} // namespace hvvr
