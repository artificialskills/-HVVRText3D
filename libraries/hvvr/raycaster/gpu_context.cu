/**
 * Copyright (c) 2017-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree. An additional grant
 * of patent rights can be found in the PATENTS file in the same directory.
 */

#include "gpu_context.h"
#include "memory_helpers.h"

#include <hip/hip_runtime_api.h>


namespace hvvr {

bool GPUContext::cudaInit() {
    int deviceCount = 0;
    cutilSafeCall(hipGetDeviceCount(&deviceCount));

    int device = 0;
#if OUTPUT_MODE == OUTPUT_MODE_3D_API
    hipDeviceProp_t deviceProps = {};

    // if we're on Windows, search for a non-TCC device
    for (int n = 0; n < deviceCount; n++) {
        hipGetDeviceProperties(&deviceProps, n);
        if (deviceProps.tccDriver == 0) {
            device = n;
            break;
        }
    }
#endif
    cutilSafeCall(hipSetDevice(device));

    uint32_t deviceFlags = 0;
    deviceFlags |= hipDeviceMapHost;
    if (hipSuccess != hipSetDeviceFlags(deviceFlags)) {
        assert(false);
        return false;
    }

    return true;
}

void GPUContext::cudaCleanup() {
    cutilSafeCall(hipProfilerStop()); // Flush profiling data for nvprof
}

GPUContext::GPUContext() : graphicsResourcesMapped(false) {}

GPUContext::~GPUContext() {
    cleanup();
}

void GPUContext::getCudaGraphicsResources(std::vector<hipGraphicsResource_t>& resources) {
    for (const auto& c : cameras) {
        if (c->resultsResource) {
            resources.push_back(c->resultsResource);
        }
    }
}

void GPUContext::interopMapResources() {
    if (!graphicsResourcesMapped) {
        std::vector<hipGraphicsResource_t> resources;
        getCudaGraphicsResources(resources);
        hipStream_t stream = 0;
        if (resources.size() > 0) {
            cutilSafeCall(hipGraphicsMapResources((int)resources.size(), resources.data(), stream));
        }

        for (auto& c : cameras) {
            // Assumes if the result image is a linear vector, that we are directly writing into the result resource
            if (c->resultImage.height() <= 1) {
                if (c->resultsResource) {
                    c->resultImage.updateFromLinearGraphicsResource(c->resultsResource, c->d_sampleRemap.size(),
                                                                    outputModeToPixelFormat(c->outputMode));
                }
            }
        }

        graphicsResourcesMapped = true;
    }
}

void GPUContext::interopUnmapResources() {
    if (graphicsResourcesMapped) {
        std::vector<hipGraphicsResource_t> resources;
        getCudaGraphicsResources(resources);
        hipStream_t stream = 0;
        if (resources.size() > 0) {
            cutilSafeCall(hipGraphicsUnmapResources((int)resources.size(), resources.data(), stream));
        }
        graphicsResourcesMapped = false;
    }
}

void GPUContext::cleanup() {
    interopUnmapResources();
    for (auto& c : cameras) {
        if (c->resultsResource) {
            cutilSafeCall(hipGraphicsUnregisterResource(c->resultsResource));
        }
        c->resultImage.reset();
        c->d_sampleResults = GPUBuffer<uint32_t>();

        safeCudaEventDestroy(c->transferTileToCPUEvent);
        safeCudaStreamDestroy(c->stream);
        safeCudaFreeHost(c->tileFrustaPinned);
        safeCudaFreeHost(c->cullBlockFrustaPinned);
        safeCudaFreeHost(c->foveatedWorldSpaceTileFrustaPinned);
        safeCudaFreeHost(c->foveatedWorldSpaceBlockFrustaPinned);
    }
    cameras.clear();
}

GPUCamera* GPUContext::getCreateCamera(const Camera* cameraPtr, bool& created) {
    created = false;
    for (size_t i = 0; i < cameras.size(); ++i) {
        if (cameras[i]->cameraPtr == cameraPtr) {
            return cameras[i].get();
        }
    }

    cameras.emplace_back(std::make_unique<GPUCamera>(cameraPtr));

    created = true;
    return (cameras.end() - 1)->get();
}

} // namespace hvvr
